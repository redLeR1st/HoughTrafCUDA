
#include <hip/hip_runtime.h>



// Accum
#define M_PI 3.14159265358979323846   // pi
#define DEG2RAD (M_PI/180.0f)
__global__ void computeAccum(unsigned char* result, unsigned char* bw_image, unsigned int* accum, int w, int h, int w_accum, int h_accum, double hough_h)
{
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int y = blockDim.y * blockIdx.y + threadIdx.y;

	int tid = y * w + x;
    int _r = w * h * 0;
    int _g = w * h * 1;
    int _b = w * h * 2;
    int _a = w * h * 3;	

	if (x >= w || y >= h)
		return;

    double center_x = w / 2;
    double center_y = h / 2;

	if (result[tid + _r] > 128 && result[tid + _g] > 128 && result[tid + _b] > 128) {
		
		for (int t = 0; t < 180; t++) {
			double r = (((double)x - center_x) * cos((double)t * DEG2RAD)) + (((double)y - center_y) * sin((double)t * DEG2RAD));
			//accum[(int)((round(r + hough_h) * 180.0)) + t]++;
            int help = (int)((round(r + hough_h) * 180.0)) + t;
            int ind = 0;
            int alma = help / h_accum;
            int korte = help % h_accum;

            ind = alma + (korte * w_accum);
            atomicAdd(&accum[(int)((round(r + hough_h) * 180.0)) + t], 1);
            //atomicAdd(&accum[ind], 1);
		}
		bw_image[tid + _r] = 255;
		bw_image[tid + _g] = 255;
        bw_image[tid + _b] = 255;
		bw_image[tid + _a] = 255;
	}
	else {
		bw_image[tid + _r] = 0;
		bw_image[tid + _g] = 0;
        bw_image[tid + _b] = 0;
		bw_image[tid + _a] = 255;
	}
	

	return;
}

