
#include <hip/hip_runtime.h>

// draw ACC
__global__ void drawAccum(unsigned int* accum, unsigned char* image_accum, int w_accum, int h_accum, int* max) {
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int y = blockDim.y * blockIdx.y + threadIdx.y;

	int tid = y * w_accum + x;

    int _r = w_accum * h_accum * 0;
    int _g = w_accum * h_accum * 1;
    int _b = w_accum * h_accum * 2;
    int _a = w_accum * h_accum * 3;	

	if (x >= w_accum || y >= h_accum)
		return;
	int pixel_value = 0;
	if (!accum[tid] == 0) {
		image_accum[tid + _r] = (unsigned char)((*max) / accum[tid]) * 255;
		image_accum[tid + _g] = (unsigned char)((*max) / accum[tid]) * 255;
		image_accum[tid + _b] = (unsigned char)((*max) / accum[tid]) * 255;
		image_accum[tid + _a] = 255;
	}
	else if (accum[tid] == -1) {
		image_accum[tid + _r] = 255;
		image_accum[tid + _g] = (unsigned char)((*max) / accum[tid]) * 255;
		image_accum[tid + _b] = (unsigned char)((*max) / accum[tid]) * 255;
		image_accum[tid + _a] = 255;
	}
	else {
		image_accum[tid + _r] = 0;
		image_accum[tid + _g] = 0;
		image_accum[tid + _b] = 0;
		image_accum[tid + _a] = 255;
	}

}