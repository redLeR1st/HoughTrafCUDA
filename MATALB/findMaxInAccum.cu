
#include <hip/hip_runtime.h>

//MAx
__global__ void findMaxInAccum(unsigned int* accum, int w_accum, int h_accum, int* dev_points, int* max)
{

	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int y = blockDim.y * blockIdx.y + threadIdx.y;

	int tid = y * w_accum + x;

	if (x >= w_accum || y >= h_accum)
		return;

	int old = (int)accum[tid];
	atomicMax(&max[0], (int)accum[tid]);

	if (old == max[0]) {
	
		atomicExch(&dev_points[0], x);
		atomicExch(&dev_points[1], y);

	}

	return;
}