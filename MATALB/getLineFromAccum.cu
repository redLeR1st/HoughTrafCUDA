
#include <hip/hip_runtime.h>
__global__ void getLineFromAccum(unsigned int* accum, int w_accum, int h_accum, int* dev_points, int* max) {
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int y = blockDim.y * blockIdx.y + threadIdx.y;

	int tid = y * w_accum + x;

	if (x >= w_accum || y >= h_accum)
		return;

	int temp_max;
	if (max[0] == (int)accum[tid]) {
		dev_points[0] = x;
		dev_points[1] = y;
		// DELETE THE LINE FROM ACCU
		
		int filter_size = 30;
		for (int i = x - filter_size / 2; i < x + filter_size / 2; i++) {
			for (int j = y - filter_size / 2; j < y + filter_size / 2; j++) {
				if (i < w_accum && j < h_accum) {
					int tid_temp = j * w_accum + i;
					accum[tid_temp] = 0;
				}
			}
		}

	}

	return;
}