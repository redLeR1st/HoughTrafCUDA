
#include <hip/hip_runtime.h>
__global__ void plotLines(unsigned char* result, int w, int h, int* points, unsigned char blue, unsigned char green, unsigned char read) {

	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int y = blockDim.y * blockIdx.y + threadIdx.y;

	int tid = y * w + x;
    
    int _r = w * h * 0;
    int _g = w * h * 1;
    int _b = w * h * 2;
    int _a = w * h * 3;	

	if (x >= w || y >= h)
		return;

	int x1 = points[0];
	int y1 = points[1];
	int x2 = points[2];
	int y2 = points[3];

	//ax + by + c = 0
	int a = y1 - y2;
	int b = x2 - x1;
	int c = (x1 - x2) * y1 + (y2 - y1) * x1;

	double diff = a * x + b * y + c;

	if (-10 < diff && diff < 500) {
        result[tid + _b] = blue;
		result[tid + _g] = green;
		result[tid + _r] = read;
		result[tid + _a] = 255;
	}

	return;
}
