#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_math_constants.h"

#include "ImageIO.h"

#include <iostream>
#include <ctime>
#include <fstream>

using namespace std;

//#define N           512
#define BLOCKDIM    16


#define M_PI 3.14159265358979323846   // pi
#define DEG2RAD (M_PI/180.0f)

__global__ void computeAccum(unsigned char* result, unsigned char* bw_image, unsigned int* accum, int w, int h, int w_accum, int h_accum, double hough_h);
__global__ void findMaxInAccum(unsigned int* accum, int w_accum, int h_accum, int* dev_points, int* max);
__global__ void plotLines(unsigned char* result, int w, int h, int* points, unsigned char b, unsigned char g, unsigned char r);
__global__ void getLineFromAccum(unsigned int* accum, int w_accum, int h_accum, int* dev_points, int* max);
__global__ void drawAccum(unsigned int* accum, unsigned char* image_accum, int w_accum, int h_accum, int* max);

int main(int argc, char** argv)
{

	int number_of_lines = std::atoi(argv[2]);

	int _img_w;
	int _img_h;

	unsigned char* result_temp = new unsigned char[2048 * 2048 * 4];
	
	char* name_of_the_input = argv[1];
	//hipHostAlloc(&result_temp, sizeof(unsigned char) * 2048 * 2048 * 4, hipHostMallocMapped);
	//hipHostRegister(&result_temp, sizeof(unsigned char) * 2048 * 2048 * 4, hipHostRegisterDefault);
	// READING STUFF
	readRGBImageFromFile(name_of_the_input, result_temp, _img_w, _img_h);


	unsigned char* result = new unsigned char[_img_w * _img_h * 4];
	unsigned char* dev_result;

	unsigned char* bw_image = new unsigned char[_img_w * _img_h * 4];
	unsigned char* dev_bw_image;

	int w_accum = 180;

	int N = _img_h > _img_w ? _img_h : _img_w;

	double hough_h = ((sqrt(2.0) * (double)N) / 2.0);
	int h_accum = hough_h * 2.0; 

	unsigned int* accum = new unsigned int[w_accum * h_accum];
	unsigned int* dev_accum;
	unsigned char* image_accum = new unsigned char[w_accum * h_accum * 4];
	unsigned char* dev_image_accum;


	int* points = new int[4];
	int* dev_points;

	int* max = new int;
	int* dev_max;

	// OPTIMIZATION######################################
	

	//hipHostAlloc(&result, sizeof(unsigned char) * _img_w * _img_h * 4, hipHostMallocMapped);
	//hipHostRegister(&result, sizeof(unsigned char) * _img_w * _img_h * 4, hipHostRegisterDefault);
	//
	//hipHostAlloc(&bw_image, sizeof(unsigned char) * _img_w * _img_h * 4, hipHostMallocMapped);
	//hipHostRegister(&bw_image, sizeof(unsigned char) * _img_w * _img_h * 4, hipHostRegisterDefault);
	//
	//hipHostAlloc(&accum, sizeof(unsigned int) * w_accum * h_accum, hipHostMallocMapped);
	//hipHostRegister(&accum, sizeof(unsigned int) * w_accum * h_accum, hipHostRegisterDefault);
	//
	//hipHostAlloc(&image_accum, sizeof(unsigned char) * w_accum * h_accum * 4, hipHostMallocMapped);
	//hipHostRegister(&image_accum, sizeof(unsigned char) * w_accum * h_accum * 4, hipHostRegisterDefault);
	//
	//hipHostAlloc(&points, sizeof(int) * 4, hipHostMallocMapped);
	//hipHostRegister(&points, sizeof(int) * 4, hipHostRegisterDefault);
	//
	//hipHostAlloc(&max, sizeof(int), hipHostMallocMapped);
	//hipHostRegister(&max, sizeof(int), hipHostRegisterDefault);
	
	// OPTIMIZATION######################################*/
	

	hipMalloc((void**)&dev_result, _img_h * _img_w * 4 * sizeof(unsigned char));

	hipMalloc((void**)&dev_bw_image, _img_h * _img_w * 4 * sizeof(unsigned char));

	hipMalloc((void**)&dev_accum, w_accum * h_accum * sizeof(unsigned int));
	hipMemset(dev_accum, 0, w_accum * h_accum * sizeof(unsigned int));

	hipMalloc((void**)&dev_image_accum, w_accum * h_accum * 4 * sizeof(unsigned char));

	hipMalloc((void**)&dev_points, 4 * sizeof(int));
	hipMemset(dev_points, 0, 4 * sizeof(int));

	hipMalloc((void**)&dev_max, sizeof(int));

	dim3 blockDim = dim3(BLOCKDIM, BLOCKDIM, 1);
	dim3 gridDim = dim3((N + BLOCKDIM - 1) / BLOCKDIM, (N + BLOCKDIM - 1) / BLOCKDIM, 1);

	std::cout << "BLOCK: " << BLOCKDIM << std::endl;
	std::cout << "GRID : " << (N + BLOCKDIM - 1) / BLOCKDIM << std::endl;
	
	hipMemcpy(dev_result, result_temp, _img_w * _img_h * 4 * sizeof(unsigned char), hipMemcpyHostToDevice);

	unsigned char b = 50;
	unsigned char g = 50;
	unsigned char r = 255;

	// ALGORITHM
	int threshlod = 20;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);


	hipEventRecord(start); //-------------------------------------------START

	computeAccum << <gridDim, blockDim >> > (dev_result, dev_bw_image, dev_accum, _img_w, _img_h, w_accum, h_accum, hough_h); // count the accum image
	for (int i = 0; i < number_of_lines; i++) {
		hipMemset(dev_max, -999999, sizeof(int));
		findMaxInAccum << <gridDim, blockDim >> > (dev_accum, w_accum, h_accum, dev_points, dev_max);

		//if (i == 0) {
		//	drawAccum << <gridDim, blockDim >> > (dev_accum, dev_image_accum, w_accum, h_accum, dev_max);
		//	hipMemcpy(image_accum, dev_image_accum, w_accum * h_accum * 4 * sizeof(unsigned char), hipMemcpyDeviceToHost);
		//	char* accum_name = "0_accum.png";
		//	accum_name[0] = '0' + i;
		//	writeRGBImageToFile(accum_name, image_accum, w_accum, h_accum);
		//}
		
		getLineFromAccum << <gridDim, blockDim >> > (dev_accum, w_accum, h_accum, dev_points, dev_max);

		hipMemcpy(max, dev_max, sizeof(int), hipMemcpyDeviceToHost);
		std::cout << "max: " << (*max) << std::endl;

		if ((*max) < threshlod) {
			std::cout << "no more line above the threshold: " << threshlod << std::endl;
			break;
		}

		hipMemcpy(points, dev_points, 4 * sizeof(int), hipMemcpyDeviceToHost);

		int x1, y1, x2, y2;
		x1 = y1 = x2 = y2 = 0;
		int x = points[0];
		int y = points[1];

		std::cout << "x " << x << std::endl;
		std::cout << "y " << y << std::endl;

		if (x >= 45 && x <= 135)
		{
			//y = (r - x cos(t)) / sin(t)  
			x1 = 0;
			y1 = ((double)(y - (h_accum / 2)) - ((x1 - (_img_w / 2)) * cos(x * DEG2RAD))) / sin(x * DEG2RAD) + (_img_h / 2);
			x2 = _img_w - 0;
			y2 = ((double)(y - (h_accum / 2)) - ((x2 - (_img_w / 2)) * cos(x * DEG2RAD))) / sin(x * DEG2RAD) + (_img_h / 2);
		}
		else
		{
			//x = (r - y sin(t)) / cos(t);  
			y1 = 0;
			x1 = ((double)(y - (h_accum / 2)) - ((y1 - (_img_h / 2)) * sin(x * DEG2RAD))) / cos(x * DEG2RAD) + (_img_w / 2);
			y2 = _img_h - 0;
			x2 = ((double)(y - (h_accum / 2)) - ((y2 - (_img_h / 2)) * sin(x * DEG2RAD))) / cos(x * DEG2RAD) + (_img_w / 2);
		}
		points[0] = x1;
		points[1] = y1;
		points[2] = x2;
		points[3] = y2;

		if (b + 30 < 255 && r - 20 >= 0) {
			b += 30;
			r -= 20;
		}

		// hipMemcpy(dev_points, points, 4 * sizeof(int), hipMemcpyHostToDevice);
		// plotLines << <gridDim, blockDim >> > (dev_result, _img_w, _img_h, dev_points, b, g, r);
		// plotLines << <gridDim, blockDim >> > (dev_bw_image, _img_w, _img_h, dev_points, b, g, r);

		

		for (int i = 0; i < 4; i++) {
			std::cout << points[i] << " ";
		}
		std::cout << "\n";
	}

	hipEventRecord(stop); // ---------------------------------------------STOP
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);

	// WRITE OUT STUFF
	hipMemcpy(result, dev_result, _img_h * _img_w * 4 * sizeof(unsigned char), hipMemcpyDeviceToHost);
	writeRGBImageToFile("result.png", result, _img_w, _img_h);

	// WRITE OUT STUFF
	hipMemcpy(bw_image, dev_bw_image, _img_h * _img_w * 4 * sizeof(unsigned char), hipMemcpyDeviceToHost);
	writeRGBImageToFile("result_bw.png", bw_image, _img_w, _img_h);

	// write accum to a file
	hipMemcpy(accum, dev_accum, w_accum * h_accum * sizeof(unsigned int), hipMemcpyDeviceToHost);
	fstream myfile;
	
	myfile.open("example.txt", fstream::out);
	
	for (int i = 0; i < N; i++) //This variable is for each row below the x 
	{
		for (int j = 0; j < 180; j++)
		{
			
			int tid = i * 180 + j;

			myfile << accum[tid] << " ";
		}
		myfile<< ";"<< std::endl;
	}
	myfile.close();
	cout << "TIME: " << milliseconds << endl;
	cout << "SIZE: " << N << endl;
	cout << "hough: " << hough_h << endl;

	return 0;
}

__global__ void plotLines(unsigned char* result, int w, int h, int* points, unsigned char blue, unsigned char green, unsigned char read) {

	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int y = blockDim.y * blockIdx.y + threadIdx.y;

	int tid = y * w + x;
	
	if (x >= w || y >= h)
		return;

	int x1 = points[0];
	int y1 = points[1];
	int x2 = points[2];
	int y2 = points[3];

	//ax + by + c = 0
	int a = y1 - y2;
	int b = x2 - x1;
	int c = (x1 - x2) * y1 + (y2 - y1) * x1;

	double diff = a * x + b * y + c;

	if (-10 < diff && diff < 500) {
		result[tid * 4 + 0] = blue;
		result[tid * 4 + 1] = green;
		result[tid * 4 + 2] = read;
		result[tid * 4 + 3] = 255;
	}

	return;
}


__global__ void getLineFromAccum(unsigned int* accum, int w_accum, int h_accum, int* dev_points, int* max) {
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int y = blockDim.y * blockIdx.y + threadIdx.y;

	int tid = y * w_accum + x;

	if (x >= w_accum || y >= h_accum)
		return;

	int temp_max;
	if (max[0] == (int)accum[tid]) {
		dev_points[0] = x;
		dev_points[1] = y;
		// DELETE THE LINE FROM ACCU
		
		int filter_size = 30;
		for (int i = x - filter_size / 2; i < x + filter_size / 2; i++) {
			for (int j = y - filter_size / 2; j < y + filter_size / 2; j++) {
				if (i < w_accum && j < h_accum) {
					int tid_temp = j * w_accum + i;
					accum[tid_temp] = -1;
				}
			}
		}

	}

	return;
}

__global__ void findMaxInAccum(unsigned int* accum, int w_accum, int h_accum, int* dev_points, int* max)
{

	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int y = blockDim.y * blockIdx.y + threadIdx.y;

	int tid = y * w_accum + x;

	if (x >= w_accum || y >= h_accum)
		return;

	int old = (int)accum[tid];
	atomicMax(&max[0], (int)accum[tid]);

	if (old == max[0]) {
	
		atomicExch(&dev_points[0], x);
		atomicExch(&dev_points[1], y);

	}

	return;
}

__global__ void computeAccum(unsigned char* result, unsigned char* bw_image, unsigned int* accum, int w, int h, int w_accum, int h_accum, double hough_h)
{
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int y = blockDim.y * blockIdx.y + threadIdx.y;

	int tid = y * w + x;
	
	if (x >= w || y >= h)
		return;

    double center_x = w / 2;
    double center_y = h / 2;

	if (result[tid * 4] > 128 && result[tid * 4 + 1]  > 128 && result[tid * 4 + 2] > 128) {
		
		for (int t = 0; t < 180; t++) {
			double r = (((double)x - center_x) * cos((double)t * DEG2RAD)) + (((double)y - center_y) * sin((double)t * DEG2RAD));
			//accum[(int)((round(r + hough_h) * 180.0)) + t]++;
			atomicAdd(&accum[(int)((round(r + hough_h) * 180.0)) + t], 1);
		}
		bw_image[tid * 4 + 0] = 255;
		bw_image[tid * 4 + 1] = 255;
		bw_image[tid * 4 + 2] = 255;
		bw_image[tid * 4 + 3] = 255;
	}
	else {
		bw_image[tid * 4 + 0] = 0;
		bw_image[tid * 4 + 1] = 0;
		bw_image[tid * 4 + 2] = 0;
		bw_image[tid * 4 + 3] = 255;
	}
	

	return;
}

__global__ void drawAccum(unsigned int* accum, unsigned char* image_accum, int w_accum, int h_accum, int* max) {
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int y = blockDim.y * blockIdx.y + threadIdx.y;

	int tid = y * w_accum + x;

	if (x >= w_accum || y >= h_accum)
		return;
	int pixel_value = 0;
	if (!accum[tid] == 0) {
		image_accum[tid * 4 + 0] = (unsigned char)((*max) / accum[tid]) * 255;
		image_accum[tid * 4 + 1] = (unsigned char)((*max) / accum[tid]) * 255;
		image_accum[tid * 4 + 2] = (unsigned char)((*max) / accum[tid]) * 255;
		image_accum[tid * 4 + 3] = 255;
	}
	else if (accum[tid] == -1) {
		image_accum[tid * 4 + 0] = 255;
		image_accum[tid * 4 + 1] = (unsigned char)((*max) / accum[tid]) * 255;
		image_accum[tid * 4 + 2] = (unsigned char)((*max) / accum[tid]) * 255;
		image_accum[tid * 4 + 3] = 255;
	}
	else {
		image_accum[tid * 4 + 0] = 0;
		image_accum[tid * 4 + 1] = 0;
		image_accum[tid * 4 + 2] = 0;
		image_accum[tid * 4 + 3] = 255;
	}

}