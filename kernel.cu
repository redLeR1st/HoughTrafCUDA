#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_math_constants.h"

#include "ImageIO.h"

#include <iostream>
#include <ctime>
#include <fstream>

using namespace std;

//#define N           512
#define BLOCKDIM    16


#define M_PI 3.14159265358979323846   // pi
#define DEG2RAD (M_PI/180.0f)

__global__ void rippleKernel(unsigned char* result, unsigned int* accum, int w, int h, int w_accum, int h_accum, double hough_h);
__global__ void getLines(unsigned int* accum, int w_accum, int h_accum, int* dev_points, int* max);
__global__ void plotLines(unsigned char* result, int w, int h, int* points);
__global__ void findMax(unsigned int* accum, int w_accum, int h_accum, int* dev_points, int* max);

int main(int argc, char** argv)
{

	int _img_w;
	int _img_h;

	unsigned char* result_temp = new unsigned char[2048 * 2048 * 4];

	std::cout << "DEBUG0" << std::endl;
	
	char* name_of_the_input = argv[1];
	// READING STUFF
	readRGBImageFromFile(name_of_the_input, result_temp, _img_w, _img_h);


	unsigned char* result = new unsigned char[_img_w * _img_h * 4];


	int N = _img_h > _img_w ? _img_h : _img_w;

	unsigned char* dev_result;

	int w_accum = 180;

	double hough_h = ((sqrt(2.0) * (double)N) / 2.0);
	int h_accum = hough_h * 2.0; // -r -> +r 

	unsigned int* accum = new unsigned int[w_accum * h_accum];
	unsigned int* dev_accum;

	int* points = new int[4];
	int* dev_points;

	int* max = new int;
	int* dev_max;

	hipMalloc((void**)&dev_result, _img_h * _img_w * 4 * sizeof(unsigned char));

	hipMalloc((void**)&dev_accum, w_accum * h_accum * sizeof(unsigned int));
	hipMemset(dev_accum, 0, w_accum * h_accum * sizeof(unsigned int));

	hipMalloc((void**)&dev_points, 4 * sizeof(int));
	hipMemset(dev_points, 0, 4 * sizeof(int));

	hipMalloc((void**)&dev_max, sizeof(int));
	hipMemset(dev_max, -999999, sizeof(int));

	dim3 blockDim = dim3(BLOCKDIM, BLOCKDIM, 1);
	dim3 gridDim = dim3((N + BLOCKDIM - 1) / BLOCKDIM, (N + BLOCKDIM - 1) / BLOCKDIM, 1);

	
	
	hipMemcpy(dev_result, result_temp, _img_w * _img_h * 4 * sizeof(unsigned char), hipMemcpyHostToDevice);
	std::cout << "DEBUG1" << std::endl;
	// ALGORITHM
	rippleKernel << <gridDim, blockDim >> > (dev_result, dev_accum, _img_w, _img_h, w_accum, h_accum, hough_h);
	getLines << <gridDim, blockDim >> > (dev_accum, w_accum, h_accum, dev_points, dev_max);
	findMax << <gridDim, blockDim >> > (dev_accum, w_accum, h_accum, dev_points, dev_max);

	hipMemcpy(points, dev_points, 4 * sizeof(int), hipMemcpyDeviceToHost);

	int x1, y1, x2, y2;
	x1 = y1 = x2 = y2 = 0;
	int x = points[0];
	int y = points[1];

	if (x >= 45 && x <= 135)
	{
		//y = (r - x cos(t)) / sin(t)  
		x1 = 0;
		y1 = ((double)(y - (h_accum / 2)) - ((x1 - (_img_w / 2)) * cos(x * DEG2RAD))) / sin(x * DEG2RAD) + (_img_h / 2);
		x2 = _img_w - 0;
		y2 = ((double)(y - (h_accum / 2)) - ((x2 - (_img_w / 2)) * cos(x * DEG2RAD))) / sin(x * DEG2RAD) + (_img_h / 2);
	}
	else
	{
		//x = (r - y sin(t)) / cos(t);  
		y1 = 0;
		x1 = ((double)(y - (h_accum / 2)) - ((y1 - (_img_h / 2)) * sin(x * DEG2RAD))) / cos(x * DEG2RAD) + (_img_w / 2);
		y2 = _img_h - 0;
		x2 = ((double)(y - (h_accum / 2)) - ((y2 - (_img_h / 2)) * sin(x * DEG2RAD))) / cos(x * DEG2RAD) + (_img_w / 2);
	}
	points[0] = x1;
	points[1] = y1;
	points[2] = x2;
	points[3] = y2;

	hipMemcpy(dev_points, points, 4 * sizeof(int), hipMemcpyHostToDevice);
	plotLines << <gridDim, blockDim >> > (dev_result, _img_w, _img_h, dev_points);

	// WRITE OUT STUFF
	hipMemcpy(result, dev_result, _img_h * _img_w * 4 * sizeof(unsigned char), hipMemcpyDeviceToHost);
	writeRGBImageToFile("result.png", result, _img_w, _img_h);
	
	hipMemcpy(max, dev_max, sizeof(int), hipMemcpyDeviceToHost);
	
	std::cout << "max: " << (*max) << std::endl;

	// write accum to a file
	hipMemcpy(accum, dev_accum, w_accum * h_accum * sizeof(unsigned int), hipMemcpyDeviceToHost);
	fstream myfile;
	
	myfile.open("example.txt", fstream::out);
	
	for (int i = 0; i < N; i++) //This variable is for each row below the x 
	{
		for (int j = 0; j < 180; j++)
		{
			
			int tid = i * 180 + j;

			myfile << accum[tid] << " ";
		}
		myfile<< ";"<< std::endl;
	}
	myfile.close();
	// ------------------



	for (int i = 0; i < 4; i++) {
		std::cout << points[i] << " ";
	}
	std::cout << "\n";

	//hipMemcpy(accum, dev_accum, w_accum * h_accum * sizeof(unsigned char), hipMemcpyDeviceToHost);
	//writeRGBImageToFile("accum.png", accum, w_accum, h_accum);

	//writeRGBImageToFile("result.png", result, N, N);
	

	return 0;
}

__global__ void plotLines(unsigned char* result, int w, int h, int* points) {

	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int y = blockDim.y * blockIdx.y + threadIdx.y;

	int tid = y * w + x;
	
	if (x >= w || y >= h)
		return;

	int x1 = points[0];
	int y1 = points[1];
	int x2 = points[2];
	int y2 = points[3];

	//ax + by + c = 0
	int a = y1 - y2;
	int b = x2 - x1;
	int c = (x1 - x2) * y1 + (y2 - y1) * x1;

	double diff = a * x + b * y + c;

	if (-10 < diff && diff < 500) {
		result[tid * 4 + 0] = 255;
		result[tid * 4 + 1] = 0;
		result[tid * 4 + 2] = 0;
		result[tid * 4 + 3] = 255;
	}

	return;
}


__global__ void findMax(unsigned int* accum, int w_accum, int h_accum, int* dev_points, int* max) {
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int y = blockDim.y * blockIdx.y + threadIdx.y;

	int tid = y * w_accum + x;

	if (x >= w_accum || y >= h_accum)
		return;

	int temp_max;
	if (max[0] == (int)accum[tid]) {
		atomicExch(&dev_points[0], x);
		atomicExch(&dev_points[1], y);
	}
	
	//atomicExch(&temp_max, max[0]);
	//
	//atomicCAS(&max[0], (int)accum[tid], x);
	//atomicExch(&dev_points[0], max[0]);
	//atomicExch(&max[0], temp_max);
	//
	//atomicCAS(&max[0], (int)accum[tid], y);
	//atomicExch(&dev_points[1], max[0]);
	//atomicExch(&max[0], temp_max);

	return;
}

__global__ void getLines(unsigned int* accum, int w_accum, int h_accum, int* dev_points, int* max)
{

	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int y = blockDim.y * blockIdx.y + threadIdx.y;

	int tid = y * w_accum + x;

	if (x >= w_accum || y >= h_accum)
		return;

	int old = (int)accum[tid];
	atomicMax(&max[0], (int)accum[tid]);

	if (old == max[0]) {
	
	
		atomicExch(&dev_points[0], x);
		atomicExch(&dev_points[1], y);
	
		//dev_points[0] = x1;
		//dev_points[1] = y1;
		//dev_points[2] = x2;
		//dev_points[3] = y2;
	}

	return;
}

__global__ void rippleKernel(unsigned char* result, unsigned int* accum, int w, int h, int w_accum, int h_accum, double hough_h)
{
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int y = blockDim.y * blockIdx.y + threadIdx.y;

	int tid = y * w + x;
	
	if (x >= w || y >= h)
		return;

    double center_x = w / 2;
    double center_y = h / 2;

	if (result[tid * 4] > 128 && result[tid * 4 + 1]  > 128 && result[tid * 4 + 2] > 128 && result[tid * 4 + 3] > 128) {
		
		for (int t = 0; t < 180; t++) {
			double r = (((double)x - center_x) * cos((double)t * DEG2RAD)) + (((double)y - center_y) * sin((double)t * DEG2RAD));
			//accum[(int)((round(r + hough_h) * 180.0)) + t]++;
			atomicAdd(&accum[(int)((round(r + hough_h) * 180.0)) + t], 1);
		}

		
		//result[tid * 4 + 1] = 0;
		//result[tid * 4 + 2] = 0;
		//result[tid * 4 + 3] = 255;
	}
	
	/*
	float dist = sqrtf((x - w / 2.0f) * (x - w / 2.0f) + (y - h / 2.0f) * (y - h / 2.0f));
	float value = (cosf(dist / waveLength * HIP_PI_F * 2) + 1) * 127;

	if (x < w && y < h)
	{
		result[tid * 4] = value;
		result[tid * 4 + 1] = value;
		result[tid * 4 + 2] = value;
		result[tid * 4 + 3] = 255;
	}
	*/
	return;
}




















/*#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

int main()
{
    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
*/